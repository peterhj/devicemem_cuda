/*
Copyright 2016-2017 the devicemem_cuda authors

Licensed under the Apache License, Version 2.0 (the "License");
you may not use this file except in compliance with the License.
You may obtain a copy of the License at

    http://www.apache.org/licenses/LICENSE-2.0

Unless required by applicable law or agreed to in writing, software
distributed under the License is distributed on an "AS IS" BASIS,
WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
See the License for the specific language governing permissions and
limitations under the License.
*/


#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>
// FIXME(20160123): commentng out for cuda 7.0.
//#include <cuda_fp16.h>

#include <stdint.h>

__global__ void vector_set_scalar_f32_kernel(
    float *dst,
    int dim,
    float c)
{
  int idx = threadIdx.x + blockIdx.x * blockDim.x;
  if (idx < dim) {
    dst[idx] = c;
  }
}

extern "C" void devicemem_cuda_vector_set_scalar_f32(
    float *dst,
    size_t dim,
    float c,
    hipStream_t stream)
{
  vector_set_scalar_f32_kernel<<<(dim+1024-1)/1024, 1024, 0, stream>>>(
      dst, dim, c);
}

__global__ void vector_add_constant_f32_kernel(
    float *dst,
    int dim,
    float c)
{
  int idx = threadIdx.x + blockIdx.x * blockDim.x;
  if (idx < dim) {
    float y = dst[idx] + c;
    dst[idx] = y;
  }
}

extern "C" void devicemem_cuda_vector_add_constant_f32(
    float *dst,
    size_t dim,
    float c,
    hipStream_t stream)
{
  vector_add_constant_f32_kernel<<<(dim+1024-1)/1024, 1024, 0, stream>>>(
      dst, dim, c);
}

__global__ void vector_add_scalar_f32_kernel(
    uint32_t dim,
    const float *c,
    float *y)
{
  uint32_t idx = threadIdx.x + blockIdx.x * blockDim.x;
  if (idx < dim) {
    y[idx] += c[0];
  }
}

extern "C" void devicemem_cuda_vector_add_scalar_f32(
    size_t dim,
    const float *c,
    float *y,
    hipStream_t stream)
{
  vector_add_scalar_f32_kernel<<<(dim+1024-1)/1024, 1024, 0, stream>>>(
      dim, c, y);
}

__global__ void vector_scale_f32_kernel(
    float *dst,
    int dim,
    float alpha)
{
  int idx = threadIdx.x + blockIdx.x * blockDim.x;
  if (idx < dim) {
    float y = alpha * dst[idx];
    dst[idx] = y;
  }
}

extern "C" void devicemem_cuda_vector_scale_f32(
    float *dst,
    size_t dim,
    float alpha,
    hipStream_t stream)
{
  vector_scale_f32_kernel<<<(dim+1024-1)/1024, 1024, 0, stream>>>(
      dst, dim, alpha);
}

__global__ void vector_div_scalar_f32_kernel(
    float *dst,
    int dim,
    float c)
{
  int idx = threadIdx.x + blockIdx.x * blockDim.x;
  if (idx < dim) {
    float y = dst[idx] / c;
    dst[idx] = y;
  }
}

extern "C" void devicemem_cuda_vector_div_scalar_f32(
    float *dst,
    size_t dim,
    float c,
    hipStream_t stream)
{
  vector_div_scalar_f32_kernel<<<(dim+1024-1)/1024, 1024, 0, stream>>>(
      dst, dim, c);
}

__global__ void vector_exp_f32_kernel(
    float *xs,
    int dim)
{
  int idx = threadIdx.x + blockIdx.x * blockDim.x;
  if (idx < dim) {
    float x = expf(xs[idx]);
    xs[idx] = x;
  }
}

extern "C" void devicemem_cuda_vector_exp_f32(
    float *xs,
    size_t dim,
    hipStream_t stream)
{
  vector_exp_f32_kernel<<<(dim+1024-1)/1024, 1024, 0, stream>>>(
      xs, dim);
}

__global__ void vector_square_f32_kernel(
    float *dst,
    int dim)
{
  int idx = threadIdx.x + blockIdx.x * blockDim.x;
  if (idx < dim) {
    float x = dst[idx];
    dst[idx] = x * x;
  }
}

extern "C" void devicemem_cuda_vector_square_f32(
    float *dst,
    size_t dim,
    hipStream_t stream)
{
  vector_square_f32_kernel<<<(dim+1024-1)/1024, 1024, 0, stream>>>(
      dst, dim);
}

__global__ void vector_reciprocal_f32_kernel(
    float *dst,
    int dim)
{
  int idx = threadIdx.x + blockIdx.x * blockDim.x;
  if (idx < dim) {
    float y = 1.0f / dst[idx];
    dst[idx] = y;
  }
}

extern "C" void devicemem_cuda_vector_reciprocal_f32(
    float *dst,
    size_t dim,
    hipStream_t stream)
{
  vector_reciprocal_f32_kernel<<<(dim+1024-1)/1024, 1024, 0, stream>>>(
      dst, dim);
}

__global__ void vector_set_f32_kernel(
    const float *src,
    int dim,
    float alpha,
    float *dst)
{
  int idx = threadIdx.x + blockIdx.x * blockDim.x;
  if (idx < dim) {
    float y = alpha * src[idx];
    dst[idx] = y;
  }
}

extern "C" void devicemem_cuda_vector_set_f32(
    const float *src,
    size_t dim,
    float alpha,
    float *dst,
    hipStream_t stream)
{
  vector_set_f32_kernel<<<(dim+1024-1)/1024, 1024, 0, stream>>>(
      src, dim, alpha, dst);
}

__global__ void vector_add_f32_kernel(
    const float *src,
    int dim,
    float alpha,
    float *dst)
{
  int idx = threadIdx.x + blockIdx.x * blockDim.x;
  if (idx < dim) {
    float y = alpha * src[idx] + dst[idx];
    dst[idx] = y;
  }
}

extern "C" void devicemem_cuda_vector_add_f32(
    const float *src,
    size_t dim,
    float alpha,
    float *dst,
    hipStream_t stream)
{
  vector_add_f32_kernel<<<(dim+1024-1)/1024, 1024, 0, stream>>>(
      src, dim, alpha, dst);
}

__global__ void vector_average_f32_kernel(
    const float *src,
    int dim,
    float alpha,
    float *dst)
{
  int idx = threadIdx.x + blockIdx.x * blockDim.x;
  if (idx < dim) {
    float y = dst[idx];
    y = y + alpha * (src[idx] - y);
    dst[idx] = y;
  }
}

extern "C" void devicemem_cuda_vector_average_f32(
    const float *src,
    size_t dim,
    float alpha,
    float *dst,
    hipStream_t stream)
{
  vector_average_f32_kernel<<<(dim+1024-1)/1024, 1024, 0, stream>>>(
      src, dim, alpha, dst);
}

__global__ void vector_elemwise_mult_f32_kernel(
    float *ys,
    int dim,
    const float *xs)
{
  int idx = threadIdx.x + blockIdx.x * blockDim.x;
  if (idx < dim) {
    float y = xs[idx] * ys[idx];
    ys[idx] = y;
  }
}

extern "C" void devicemem_cuda_vector_elemwise_mult_f32(
    float *dst,
    size_t dim,
    const float *xs,
    hipStream_t stream)
{
  vector_elemwise_mult_f32_kernel<<<(dim+1024-1)/1024, 1024, 0, stream>>>(
      dst, dim, xs);
}

__global__ void vector_elemwise_div_f32_kernel(
    float *ys,
    int dim,
    const float *xs)
{
  int idx = threadIdx.x + blockIdx.x * blockDim.x;
  if (idx < dim) {
    float y = ys[idx] / xs[idx];
    ys[idx] = y;
  }
}

extern "C" void devicemem_cuda_vector_elemwise_div_f32(
    float *dst,
    size_t dim,
    const float *xs,
    hipStream_t stream)
{
  vector_elemwise_div_f32_kernel<<<(dim+1024-1)/1024, 1024, 0, stream>>>(
      dst, dim, xs);
}
